#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdint.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <sys/stat.h>
#include <math.h>

// Enable ECB, CTR and CBC mode. Note this can be done before including aes.h or at compile-time.
// E.g. with GCC by using the -D flag: gcc -c aes.c -DCBC=0 -DCTR=1 -DECB=1
#define CBC 0
#define CTR 1
#define ECB 0

#include "aes.h"
#define THREADS_PER_BLOCK 1024

void test_xcrypt_ctr(const char* xcrypt)
{
    uint8_t key[16] = { 0x2b, 0x7e, 0x15, 0x16, 0x28, 0xae, 0xd2, 0xa6, 0xab, 0xf7, 0x15, 0x88, 0x09, 0xcf, 0x4f, 0x3c };
    
    uint8_t iv[16]  = { 0xf0, 0xf1, 0xf2, 0xf3, 0xf4, 0xf5, 0xf6, 0xf7, 0xf8, 0xf9, 0xfa, 0xfb, 0xfc, 0xfd, 0xfe, 0xff };
    
    struct AES_ctx ctx;
    
    if (strcmp(xcrypt, "encrypt")==0) {
        AES_init_ctx_iv(&ctx, key, iv);
        printf("Encryption kernel launching..\n");
    	static const char filenamer[] = "plaintext.txt";
    	static const char filenamee[] = "encrypted.txt";
    	FILE *fr = fopen(filenamer, "rb");
    	FILE *fe = fopen(filenamee, "wb");
    	if (fr == NULL || fe == NULL)
		exit(EXIT_FAILURE);
    	size_t len = 0, BufContSz;
    	ssize_t read;
    	int i;
        uint8_t *d_file;
        struct AES_ctx *d_ctx;
        hipMalloc(&d_ctx, sizeof(AES_ctx));
        hipMemcpy(d_ctx, &ctx, sizeof(AES_ctx), hipMemcpyHostToDevice);
        struct stat buffer;
        int status = stat(filenamer, &buffer);
        if (status != 0)
		printf("File size reading error");
        size_t file_size = buffer.st_size;
        uint8_t *file_data = (uint8_t *)malloc(file_size);
        uint8_t *file_dat = (uint8_t *)malloc(file_size);
        hipMalloc(&d_file, file_size);
        fread(file_data, 1, file_size, fr);
	int numThreads = (file_size+63)/64;
	int numBlocks = numThreads/1024;
	int sqrtBlocks = sqrt(numBlocks)+1;
        hipMemcpy(d_file, file_data, file_size, hipMemcpyHostToDevice);
        dim3 dimBlock(32, 32, 1);
        dim3 dimGrid(sqrtBlocks, sqrtBlocks, 1);
	AES_CTR_xcrypt_buffer<<<dimGrid, dimBlock>>>(d_ctx, d_file, 64, file_size);
        hipDeviceSynchronize();
        hipMemcpy(file_dat, d_file, file_size, hipMemcpyDeviceToHost);
        fwrite(file_dat, 1, file_size, fe);

    	fclose(fr);
    	fclose(fe);
	hipFree(d_file);
	hipFree(d_ctx);
	free(file_data);
	free(file_dat);
    }
    else {
        AES_init_ctx_iv(&ctx, key, iv);
        printf("Decryption kernel launching..\n");
	static const char filenamed[] = "decrypted.txt";
    	static const char filenamee[] = "encrypted.txt";
    	FILE *fe = fopen(filenamee, "rb");
    	FILE *fd = fopen(filenamed, "wb");
    	if (fe == NULL || fd == NULL)
		exit(EXIT_FAILURE);
    	size_t len = 0, BufContSz;
    	ssize_t read;
    	int i;
        uint8_t *de_file;
        struct AES_ctx *de_ctx;
        hipMalloc(&de_ctx, sizeof(AES_ctx));
        hipMemcpy(de_ctx, &ctx, sizeof(AES_ctx), hipMemcpyHostToDevice);
        struct stat buffer;
        int status = stat(filenamee, &buffer);
        if (status != 0)
		printf("File size reading error");
        size_t file_size = buffer.st_size;
        uint8_t *file_data = (uint8_t *)malloc(file_size);
        uint8_t *file_dat = (uint8_t *)malloc(file_size);
        hipMalloc(&de_file, file_size);
        fread(file_data, 1, file_size, fe);
	int numThreads = (file_size+63)/64;
	int numBlocks = numThreads/1024;
	int sqrtBlocks = sqrt(numBlocks)+1;
        hipMemcpy(de_file, file_data, file_size, hipMemcpyHostToDevice);
        dim3 dimBlock(32, 32, 1);
        dim3 dimGrid(sqrtBlocks, sqrtBlocks, 1);
	AES_CTR_xcrypt_buffer<<<dimGrid, dimBlock>>>(de_ctx, de_file, 64, file_size);
        hipDeviceSynchronize();
        hipMemcpy(file_dat, de_file, file_size, hipMemcpyDeviceToHost);
        fwrite(file_dat, 1, file_size, fd);

    	fclose(fe);
    	fclose(fd);
	hipFree(de_file);
	hipFree(de_ctx);
	free(file_data);
	free(file_dat);

    }
 
}

int main(void)
{
    clock_t start, end;
    double cpu_time_used;
    start = clock();
    test_xcrypt_ctr("encrypt");
    end = clock();
    cpu_time_used = ((double)(end - start))/CLOCKS_PER_SEC;
    printf("Encryption time: %fs\n", cpu_time_used);
    start = clock();
    test_xcrypt_ctr("decrypt");
    end = clock();
    cpu_time_used = ((double)(end - start))/CLOCKS_PER_SEC;
    printf("Decryption time: %fs\n", cpu_time_used);
    return 0;
}
